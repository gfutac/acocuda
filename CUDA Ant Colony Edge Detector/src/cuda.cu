#include "hip/hip_runtime.h"
#undef __SSE2__

#include <stdio.h>
#include <time.h>
#include <cv.h>
#include <highgui.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math_functions.h>

#include <map>
#include <algorithm>

#include "ant.h"
#include "cutil/cutil.h"


//#include "cutil_inline.h"

using namespace std;

#define ANTS 1024

#define error(msg) {\
			hipDeviceSynchronize();\
			hipError_t err = hipGetLastError();\
			if( hipSuccess != err) {\
				fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n", \
					msg, __FILE__, __LINE__, hipGetErrorString( err ) );\
					exit(EXIT_FAILURE);\
			}\
		}

void showImage(IplImage *img){
	cvNamedWindow("Slika", CV_WINDOW_AUTOSIZE);
	cvMoveWindow("Slika", 440, 65);
	cvShowImage("Slika", img);
	while(true) if (cvWaitKey(10) == 27) break;
	cvDestroyWindow("Slika");
}
//
///////////////////////////////////////////////////////////////////////////////////////////////
//

__device__ pixel d_imageMatrix[512][512];
__device__ float alpha = 1.5;
__device__ float beta = 3.5;
__device__ float testSum[512];


__device__ float getMax(float vals[4]){
	float max = vals[0];
	for (int i = 1; i < 4; ++i){
		max = max > vals[i] ? max : vals[i];
	}
	return max;
}

__device__ float myRand(unsigned long seed){
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	seed = seed + i;
	unsigned long next = seed * 1103515245 + 12345;
	unsigned long temp = ((unsigned)(next/65536) % 32768);
	return (float)temp/32768;
}

/**
 * Normalizira vrijednosti slike sivih razina.
 */
__global__ void setImageValues(float *img, size_t pitch, float maxValue){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float *q = (float *)((char *)img + j * pitch) + i;
	*q = (*q) / maxValue;

	d_imageMatrix[i][j].val = *q;
	d_imageMatrix[i][j].antCount = 0;
	d_imageMatrix[i][j].tao = 0.001;
}

/**
 * svakom pixelu odreduje vidljivost (heuristika) i postavlja mu susjede (eksplicitno gradi graf u memoriji,
 * veca prostorna slozenost programa, manja vremenska).
 */
__global__ void setNeighs(int height, int width){
    float tl, tm, tr;
    float ml, mr;
    float bl, bm, br;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float intens[4];
	tl = (i - 1 >= 0 && j - 1 >= 0) ? d_imageMatrix[i-1][j-1].val : d_imageMatrix[i][j].val;
	br = (i + 1 <= height && j + 1 <= width) ? d_imageMatrix[i+1][j+1].val : d_imageMatrix[i][j].val;
	tr = (i - 1 >= 0 && j + 1 <= width) ? d_imageMatrix[i-1][j+1].val : d_imageMatrix[i][j].val;
	bl = (i + 1 <= height && j - 1 >= 0) ? d_imageMatrix[i+1][j-1].val : d_imageMatrix[i][j].val;
	tm = (i - 1 >= 0) ? d_imageMatrix[i-1][j].val : d_imageMatrix[i][j].val;
	bm = (i + 1 < height) ? d_imageMatrix[i+1][j].val : d_imageMatrix[i][j].val;
	ml = (j - 1 >= 0) ? d_imageMatrix[i][j-1].val : d_imageMatrix[i][j].val;
	mr = (j + 1 < width) ? d_imageMatrix[i][j+1].val : d_imageMatrix[i][j].val;

	intens[0] = fabs(tl - br);
	intens[1] = fabs(tr - bl);
	intens[2] = fabs(ml - mr);
	intens[3] = fabs(tm - bm);

	d_imageMatrix[i][j].ni = d_imageMatrix[i][j].val * getMax(intens);

	int index = 0;
	if (i - 1 >= 0 && j - 1 >= 0) d_imageMatrix[i][j].neigh[index++] = &d_imageMatrix[i-1][j-1];
	if (i + 1 < height && j + 1 < width) d_imageMatrix[i][j].neigh[index++] = &d_imageMatrix[i+1][j+1];
	if (i - 1 >= 0 && j + 1 < width) d_imageMatrix[i][j].neigh[index++] = &d_imageMatrix[i-1][j+1];
	if (i + 1 < height && j - 1 >= 0) d_imageMatrix[i][j].neigh[index++] = &d_imageMatrix[i+1][j-1];
	if (i - 1 >= 0) d_imageMatrix[i][j].neigh[index++] = &d_imageMatrix[i-1][j];
	if (i + 1 < height) d_imageMatrix[i][j].neigh[index++] = &d_imageMatrix[i+1][j];
	if (j - 1 >= 0) d_imageMatrix[i][j].neigh[index++] = &d_imageMatrix[i][j-1];
	if (j + 1 < width) d_imageMatrix[i][j].neigh[index++] = &d_imageMatrix[i][j+1];

	d_imageMatrix[i][j].neighCount = index;
}

__global__ void setAnts(ant ants[ANTS]){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	int x = ants[i].startx;
	int y = ants[i].starty;
	//postavlja mrava na pocetnu poziciju
	ants[i].push_back(&d_imageMatrix[x][y]);
	atomicAdd(&d_imageMatrix[x][y].antCount, 1);
}

__global__ void walk(ant ants[ANTS], unsigned long seed){
	int antIndex = blockIdx.x * blockDim.x + threadIdx.x;
	pixel *admissible[8];
	int admissibleCount = 0;
	pixel *last = ants[antIndex].path.last();

	float probabilities[8];
	float probSum = 0;

	if (ants[antIndex].path.getCount() == 1){
		for (int i = 0; i < last->neighCount; ++i){
			admissible[i] = last->neigh[i];
			++admissibleCount;
		}

		for (int i = 0; i < admissibleCount; ++i){
			pixel *tmp = last->neigh[i];
			float probability = pow(tmp->tao, alpha) * pow(tmp->ni, beta);
			probabilities[i] = probability;
			probSum += probability;
		}
	}

	else if (ants[antIndex].path.getCount() > 1){
		pixel *penultimate = ants[antIndex].path.penultimate();
		for (int neighbors = 0; neighbors < last->neighCount; ++neighbors){
			if (ants[antIndex].path.contains(last->neigh[neighbors]) || last->neigh[neighbors] == penultimate) continue;
			admissible[admissibleCount++] = last->neigh[neighbors];
		}
		--admissibleCount;

		for (int i = 0; i < admissibleCount; ++i){
			pixel *tmp = admissible[i];
			float probability = pow(tmp->tao, alpha) * pow(tmp->ni, beta);
			probabilities[i] = probability;
			probSum += probability;
		}
	}

	float r = myRand(antIndex * 17 + seed) * probSum;
	float acumulatedSum = 0;
	pixel *next = 0;
	for (int i = 0; i < admissibleCount; ++i){
		acumulatedSum += probabilities[i];
		if (r < acumulatedSum) next = last->neigh[i];
	}
	if (!next){
		if (admissibleCount) next = admissible[admissibleCount];
		else {
			next = ants[antIndex].path[(int)myRand(antIndex * 31 + seed) * 32768 % ants[antIndex].path.size()];
		}
	}

	atomicAdd(&next->antCount, 1);
	ants[antIndex].push_back(next);
}

__global__ void updateTrails(){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0;

	if (d_imageMatrix[i][j].ni >= 0.08) {
		sum = d_imageMatrix[i][j].ni * d_imageMatrix[i][j].antCount;
	}
	d_imageMatrix[i][j].tao = d_imageMatrix[i][j].tao * (1 - 0.02) + sum;
	d_imageMatrix[i][j].antCount = 0;
}


int main(int argc, char *argv[]){
	IplImage *in = cvLoadImage("/home/gf43122/work/eclipse/workspace/ACO2/resources/lena512s.png", CV_LOAD_IMAGE_GRAYSCALE);
	int height = in->height;
	int width = in->width;
//	showImage(in);

	float maxValue = 0;
	float *hostImageValues = (float *)malloc(height * width * sizeof(float));
	for (int i = 0; i < height; ++i){
		for (int j = 0; j < width; ++j){
			*(hostImageValues + i * width + j) = ((uchar *)(in->imageData + i*in->widthStep))[j];
			maxValue = maxValue > *(hostImageValues + i * width + j) ? maxValue : *(hostImageValues + i * width + j);
		}
	}


	float *imageIntensityValues;
	size_t pitch;
	hipMallocPitch((void **)&imageIntensityValues, &pitch, sizeof(float) * width, height);
	hipMemcpy(imageIntensityValues, hostImageValues, sizeof(float) * width * height, hipMemcpyHostToDevice);

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(512 / threadsPerBlock.x, 512 / threadsPerBlock.y);
	setImageValues<<<numBlocks, threadsPerBlock>>>(imageIntensityValues, pitch, maxValue);
	setNeighs<<<numBlocks, threadsPerBlock>>>(height, width);

	map<pair<int, int>, bool> mapa;
	ant mravi[ANTS];
	srand((unsigned)time(NULL));
	int k = 0;
	while(k < ANTS){
		int i = rand() % height;
		int j = rand() % width;
		pair<int, int> lokacija (i, j);
		if (mapa.find(lokacija) != mapa.end()) continue;
		mapa[lokacija] = true;
		++k;
	}

	map<pair<int, int>, bool>::iterator it;
	int index = 0;
	for (it = mapa.begin(); it != mapa.end(); ++it){
		pair<int, int> p = it->first;
		int x = p.first;
		int y = p.second;
		int len = rand() % 15 + 25 + 1;
		mravi[index] = ant(len);
		mravi[index++].setStartPosition(x, y);
	}

	ant *deviceAnts;
	hipMalloc((void **)&deviceAnts, ANTS * sizeof(ant));
	hipMemcpy(deviceAnts, mravi, ANTS * sizeof(ant), hipMemcpyHostToDevice);
	setAnts<<<32, 32>>>(deviceAnts);


	for (int i = 0; i < 512; ++i){
		walk<<<32, 32>>>(deviceAnts, (unsigned)time(NULL));
		updateTrails<<<numBlocks, threadsPerBlock>>>();
	}


	pixel *slika = (pixel *)malloc(512 * 512 * sizeof(pixel));
	hipMemcpyFromSymbol(slika, HIP_SYMBOL("d_imageMatrix"), 512 * 512 * sizeof(pixel), 0, hipMemcpyDeviceToHost);

	IplImage *out = cvCreateImage(cvSize(width, height), IPL_DEPTH_8U, 1);

	float total = 0;
	for (int i = 0; i < height; ++i){
		for (int j = 0; j < width; ++j){
			total = total  + slika[i * width + j].tao;
		}
	}
	total /= (width * height);

	for (int i = 0; i < height; ++i){
		for (int j = 0; j < width; ++j){
			if (slika[i * width + j].tao >= total) cvSet2D(out, j, i, cvScalar(255,0,0,0));
		}
	}

	showImage(out);
	cvReleaseImage(&out);

	cvReleaseImage(&in);
	hipFree(imageIntensityValues);
	hipFree(deviceAnts);
	free(hostImageValues);
	free(slika);
    return 0;
}

