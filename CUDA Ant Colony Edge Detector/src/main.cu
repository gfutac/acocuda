#include "hip/hip_runtime.h"
#undef __SSE2__

#include <iostream>
#include <ctime>
#include <cv.h>
#include <highgui.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math_functions.h>

#include <map>

#include "ant.h"
#include "cutil/cutil.h"
#include "imageOp.h"

using namespace std;

#define ANTS 1024
#define WIDTH 512
#define HEIGHT 512

#define error(msg) {\
			hipDeviceSynchronize();\
			hipError_t err = hipGetLastError();\
			if( hipSuccess != err) {\
				fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n", \
					msg, __FILE__, __LINE__, hipGetErrorString( err ) );\
					exit(EXIT_FAILURE);\
			}\
		}

__device__ position deviceImage[512][512];
__device__ float broj;


texture<float, 2, hipReadModeElementType> imageValuesTexture;
texture<float, 2, hipReadModeElementType> heuristicsTexture;

__device__ int myRand(unsigned long seed){
	unsigned long next = seed * 1103515245 + 12345;
	unsigned long temp = ((unsigned)(next/65536) % 32768);
//	return (float)temp/32768;
	return temp;
}

__global__ void init(float *values, size_t pitch, float maxValue){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float *q = (float *)((char *)values + j * pitch) + i;
	*q /= maxValue;
	deviceImage[i][j].pheromone = 0.001;
	deviceImage[i][j].antCount = 0;
}

__global__ void setHeuristics(float *heuristics, int pitch){
	float tl, tm, tr;
	float ml, mr;
	float bl, bm, br;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float intens[4];
	float current = tex2D(imageValuesTexture, i, j);

	tl = (i - 1 >= 0 && j - 1 >= 0) ? tex2D(imageValuesTexture, i - 1, j - 1) : current;
	br = (i + 1 <= HEIGHT && j + 1 <= WIDTH) ? tex2D(imageValuesTexture, i - 1, j - 1) : current;
	tr = (i - 1 >= 0 && j + 1 <= WIDTH) ? tex2D(imageValuesTexture, i - 1, j + 1) : current;
	bl = (i + 1 <= HEIGHT && j - 1 >= 0) ? tex2D(imageValuesTexture, i + 1, j - 1) : current;
	tm = (i - 1 >= 0) ? tex2D(imageValuesTexture, i - 1, j) : current;
	bm = (i + 1 < HEIGHT) ? tex2D(imageValuesTexture, i + 1, j) : current;
	ml = (j - 1 >= 0) ? tex2D(imageValuesTexture, i, j - 1) : current;
	mr = (j + 1 < WIDTH) ? tex2D(imageValuesTexture, i, j + 1) : current;


	intens[0] = fabs(tl - br);
	intens[1] = fabs(tr - bl);
	intens[2] = fabs(ml - mr);
	intens[3] = fabs(tm - bm);

	float max = intens[0];
	for (int k = 1; k < 4; ++k) {
		max = max > intens[k] ? max : intens[k];
	}

	float *currentHeuristicValue = (float *)((char *)heuristics + j * pitch) + i;
	*currentHeuristicValue = current * max;

	int index = 0;
	if (i - 1 >= 0 && j - 1 >= 0) deviceImage[i][j].neigh[index++] = &deviceImage[i-1][j-1];
	if (i + 1 < HEIGHT && j + 1 < WIDTH) deviceImage[i][j].neigh[index++] = &deviceImage[i+1][j+1];
	if (i - 1 >= 0 && j + 1 < WIDTH) deviceImage[i][j].neigh[index++] = &deviceImage[i-1][j+1];
	if (i + 1 < HEIGHT && j - 1 >= 0) deviceImage[i][j].neigh[index++] = &deviceImage[i+1][j-1];
	if (i - 1 >= 0) deviceImage[i][j].neigh[index++] = &deviceImage[i-1][j];
	if (i + 1 < HEIGHT) deviceImage[i][j].neigh[index++] = &deviceImage[i+1][j];
	if (j - 1 >= 0) deviceImage[i][j].neigh[index++] = &deviceImage[i][j-1];
	if (j + 1 < WIDTH) deviceImage[i][j].neigh[index++] = &deviceImage[i][j+1];

	deviceImage[i][j].neighCount = index;
}

__device__ int indeksi[1024];
__global__ void setAnts(ant *ants, unsigned long seed){
	int antIndex = blockDim.x * blockIdx.x + threadIdx.x;
	int currentSeed = (seed + antIndex) << 5;
	int randIndex = myRand(currentSeed) % ANTS;

	int i = randIndex / 32;
	int j = randIndex % 32;
	//atomic compare and swap
	while (atomicCAS(&deviceImage[i][j].antCount, 1, 1)){
		currentSeed <<= 1;
		randIndex = myRand(currentSeed) % ANTS;
	}
	atomicAdd(&deviceImage[i][j].antCount, 1);
	indeksi[antIndex] = randIndex;
}

__global__ void test(){
	broj = tex2D(heuristicsTexture, 50,50);
}

int main(int argc, char **argv){
	string inputFileImage = "resources/lena512s.png";
	IplImage *inputIplImage = cvLoadImage(inputFileImage.c_str(), CV_LOAD_IMAGE_GRAYSCALE);

	//ucitavanje slike
	float hostImageValues[HEIGHT][WIDTH];
	float maxValue = 0;
	for (int i = 0; i < HEIGHT; i++){
		for (int j = 0; j < 512; ++j){
			hostImageValues[i][j] = cvGet2D(inputIplImage, i, j).val[0];
			maxValue = maxValue > hostImageValues[i][j] ? maxValue : hostImageValues[i][j];
		}
	}
	//kraj ucitavanja slike


	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(HEIGHT / threadsPerBlock.x, WIDTH / threadsPerBlock.y);

	//normaliziranje slike sivih razina, inicijalizacija feromonskih tragova
	float *deviceImageProperties;
	size_t pitch;

	hipMallocPitch((void **)&deviceImageProperties, &pitch, sizeof(float) * WIDTH, HEIGHT);
	hipMemcpy(deviceImageProperties, hostImageValues, sizeof(float) * HEIGHT * WIDTH, hipMemcpyHostToDevice);

	init<<<numBlocks, threadsPerBlock>>>(deviceImageProperties, pitch, maxValue);
	//kraj inicijalizacije

	//"bindanje" matrice sivih razina u memorijski dio za texture (konstanta memorija brza od globalne)
	hipArray *imageValuesArray;
	hipChannelFormatDesc cd = imageValuesTexture.channelDesc;

	hipMallocArray(&imageValuesArray, &cd, WIDTH, HEIGHT);
	hipMemcpyToArray(imageValuesArray, 0, 0, deviceImageProperties, sizeof(float) * HEIGHT * WIDTH, hipMemcpyDeviceToDevice);

	imageValuesTexture.addressMode[0] = hipAddressModeWrap;
	imageValuesTexture.addressMode[1] = hipAddressModeWrap;
	imageValuesTexture.filterMode     = hipFilterModePoint;
	imageValuesTexture.normalized     = false;

	hipBindTextureToArray(&imageValuesTexture, imageValuesArray, &cd);
	//kraj bindanja

	setHeuristics<<<numBlocks, threadsPerBlock>>>(deviceImageProperties, pitch);
	hipArray *heuristicsArray;
	cd = heuristicsTexture.channelDesc;

	hipMallocArray(&heuristicsArray, &cd, WIDTH, HEIGHT);
	hipMemcpyToArray(heuristicsArray, 0, 0, deviceImageProperties, sizeof(float) * HEIGHT * WIDTH, hipMemcpyDeviceToDevice);

	heuristicsTexture.addressMode[0] = hipAddressModeWrap;
	heuristicsTexture.addressMode[1] = hipAddressModeWrap;
	heuristicsTexture.filterMode     = hipFilterModePoint;
	heuristicsTexture.normalized     = false;

	hipBindTextureToArray(&heuristicsTexture, heuristicsArray, &cd);
	//kraj bindanja

	ant *ants;
	hipMalloc(&ants, sizeof(ant) * ANTS);

//	setAnts<<<32, 32>>>(ants, (unsigned)time(0));



//	test<<<1, 1>>>();
//	int t[1024];
//	hipMemcpyFromSymbol(t, HIP_SYMBOL("indeksi"), sizeof(float) * 1024, 0, hipMemcpyDeviceToHost);
//	for (int i = 0; i < ANTS; ++i)
//		cout << t[i] << " ";
//	cout << endl;



	cvReleaseImage(&inputIplImage);
	hipFree(deviceImageProperties);
	hipFree(ants);
	hipFreeArray(imageValuesArray);
	hipFreeArray(heuristicsArray);
	return 0;
}
